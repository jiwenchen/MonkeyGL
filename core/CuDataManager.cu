#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2022-2023 jiwenchen(cjwbeyond@hotmail.com)

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>


extern "C"
void cu_copyVolumeData( short* h_volumeData, hipExtent volumeSize, hipArray*& d_volumeArray, hipTextureObject_t& volumeTexture)
{
	if (d_volumeArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_volumeArray));
		d_volumeArray = 0;
		volumeTexture = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<short>();
	checkCudaErrors( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

	hipMemcpy3DParms copyParams = {0};
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.srcPtr   = make_hipPitchedPtr(
		(void*)h_volumeData,
		volumeSize.width*sizeof(short),
		volumeSize.width,
		volumeSize.height
	);

	checkCudaErrors( hipMemcpy3D(&copyParams) );  
	
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_volumeArray;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true;  // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear;  // linear interpolation

	texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeNormalizedFloat;
		
	checkCudaErrors( hipCreateTextureObject(&volumeTexture, &texRes, &texDescr, NULL) );
}

extern "C"
void cu_copyMaskData(unsigned char* h_maskData, hipExtent volumeSize, hipArray*& d_maskArray, hipTextureObject_t& maskTexture)
{
	if (d_maskArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_maskArray));
		d_maskArray = 0;
		maskTexture = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	checkCudaErrors( hipMalloc3DArray(&d_maskArray, &channelDesc, volumeSize) );

	hipMemcpy3DParms copyParams = {0};
	copyParams.dstArray = d_maskArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.srcPtr   = make_hipPitchedPtr(
		(void*)h_maskData,
		volumeSize.width*sizeof(unsigned char),
		volumeSize.width,
		volumeSize.height
	);

	checkCudaErrors( hipMemcpy3D(&copyParams) );  
	
	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_maskArray;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModePoint; 

	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeElementType;
		
	checkCudaErrors( hipCreateTextureObject(&maskTexture, &texRes, &texDescr, NULL) );
}

extern "C"
void cu_setTransferFunc( float* pTransferFunc, int nLenTransferFunc, hipArray*& d_transferFuncArray, hipTextureObject_t& transferFuncTexture)
{
    if (d_transferFuncArray != 0)
	{
		checkCudaErrors(hipFreeArray(d_transferFuncArray));
		d_transferFuncArray = 0;
		transferFuncTexture = 0;
	}

	hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

    checkCudaErrors(hipMallocArray( &d_transferFuncArray, &channelDesc, nLenTransferFunc, 1));
    checkCudaErrors(
        hipMemcpy2DToArray(
            d_transferFuncArray, 
            0, 
            0, 
            pTransferFunc,
            0, 
            nLenTransferFunc*sizeof(float4), 
            1,
            hipMemcpyHostToDevice
        )
    );

    texRes.res.array.array = d_transferFuncArray;

    checkCudaErrors(
        hipCreateTextureObject(&transferFuncTexture, &texRes, &texDescr, NULL)
    );
}